#include <vector>
#include <chrono>
#include <cstdio>
#include <cstdlib>
#include <algorithm>
#include <hip/hip_runtime.h>

#include <iostream>
#include <chrono>


__device__ void myfun(int* a,int n)
{
    int tid = threadIdx.x;
    int t = 0;
    for(int i = 1;i < n; i = i << 1)//从序列长度 1 到 序列长度 n/2，我们的目的是对 一个双调函数排序，所以不用对n
    {
        //判断升序还是降序 0为升 1为降 我们的目的是获得 序列长度为2，所以要除2*i
        bool order = (tid / (2*i))%2;
        //printf("%d %d \n",tid, order);       
        for(int j = i; j >= 1; j = j >> 1)
        {      
            if(((tid / j)%2) == 0) //除跳跃的步长 再取模 这是对自身的序列做排序，所以不用*2
            {            
                // 升序 & 出现 前 > 后   || 降序 & 前 < 后 并且在最后一次的时候，没用降序，所以一定要判断边界
                if ((tid + j < n) && (   ((!order) == (a[tid] > a[tid + j]))    ||   (order == (a[tid] < a[tid + j]))    ))
                {
                    t = a[tid];
                    a[tid] = a[tid + j];
                    a[tid + j] = t;
                }               
            }
            __syncthreads();
        }
    }
}



__global__ void myfun_shared(int* a,int n)
{
    int tid = threadIdx.x;
    
    __shared__ int a_share[1024];
    int t = 0,flag1 = 0, flag2 = 0;
    a_share[tid] = a[tid];
    __syncthreads();
    for(int i = 1;i < n; i = i << 1)//从序列长度 1 到 序列长度 n/2，我们的目的是对 一个双调函数排序，所以不用对n
    {
        //判断升序还是降序 0为升 1为降 我们的目的是获得 序列长度为2，所以要除2*i
//        bool order = (tid / (2*i))%2;
        flag1++;
        bool order = (tid >> flag1)%2;
        //printf("%d %d \n",tid, order);
        flag2 = flag1 - 1;       
        for(int j = i; j >= 1; j = j >> 1)
        {
            if(((tid >> flag2)%2) == 0) //除跳跃的步长 再取模 这是对自身的序列做排序，所以不用*2
            {            
                // 升序 & 出现 前 > 后   || 降序 & 前 < 后 并且在最后一次的时候，没用降序，所以一定要判断边界
                if ((tid + j < n) && (   ((!order) == (a_share[tid] > a_share[tid + j]))    ||   (order == (a_share[tid] < a_share[tid + j]))    ))
                {
                    t = a_share[tid];
                    a_share[tid] = a_share[tid + j];
                    a_share[tid + j] = t;
                }               
            }
            flag2--;
            __syncthreads();
        }
    }
    a[tid] = a_share[tid];
}





__global__ void warpfun(int* a,int* b,int n)
{
    int tid = threadIdx.x;
    
    __shared__ unsigned int a_share[1024];
    __shared__ unsigned int hash_share[1024];
    __shared__ unsigned int seq_share[1024];
    __shared__ unsigned int flow_share[1024];
    __shared__ unsigned int temp[2048];
    __shared__ unsigned int flag_share[1024];
    __shared__ unsigned int feature_share[1024];
    
    int t = 0,flag1 = 0, flag2 = 0;
    a_share[tid] = a[tid]*1024+tid;
//    a_share[tid] = a[tid];
    hash_share[tid] = a[tid];
    seq_share[tid] = tid + 1;
    flow_share[tid] = 1;//flow列
    //temp不需要初始化，只是一个双指针算法
    flag_share[tid] = 0;
    feature_share[tid] = tid;
    __syncthreads();
    for(int i = 1;i < n; i = i << 1)//从序列长度 1 到 序列长度 n/2，我们的目的是对 一个双调函数排序，所以不用对n
    {
        //判断升序还是降序 0为升 1为降 我们的目的是获得 序列长度为2，所以要除2*i
//        bool order = (tid / (2*i))%2;
        flag1++;
        bool order = (tid >> flag1)%2;
        //printf("%d %d \n",tid, order);
        flag2 = flag1 - 1;       
        for(int j = i; j >= 1; j = j >> 1)
        {
            if(((tid >> flag2)%2) == 0) //除跳跃的步长 再取模 这是对自身的序列做排序，所以不用*2
            {            
                // 升序 & 出现 前 > 后   || 降序 & 前 < 后 并且在最后一次的时候，没用降序，所以一定要判断边界
                if ((tid + j < n) && (   ((!order) == (a_share[tid] > a_share[tid + j]))    ||   (order == (a_share[tid] < a_share[tid + j]))    ))
                {
                    t = a_share[tid];
                    a_share[tid] = a_share[tid + j];
                    a_share[tid + j] = t;
                
                    t = seq_share[tid];
                    seq_share[tid] = seq_share[tid + j];
                    seq_share[tid + j] = t;

                    t = hash_share[tid];
                    hash_share[tid] = hash_share[tid + j];
                    hash_share[tid + j] = t;
                
                }               
            }
            flag2--;
            __syncthreads();
        }
    }
    
    //这里做flow列 邻居节点的处理
    if(tid > 0)
    {
        flow_share[tid] = (hash_share[tid] != hash_share[tid-1]);
    }
    temp[tid] = flow_share[tid];
    __syncthreads();

    
    
    int in = 1;
    int out = 0;
    //前缀和计算有问题
    if(tid < n)
    {
        for(int i = 1;i < n;i = i<<1)
            {
                in = 1 - in;
                out = 1 - out; 
                int index = i;
                if((tid - index) >= 0)
                {                
                    temp[tid + n * out] = temp[tid + n * in] + temp[tid - index + n * in];                    
//                    temp[tid + out] = temp[tid ] + temp[tid - index ];                
                }
                else
                {
                    temp[tid + n * out] = temp[tid + n * in];
                }
                __syncthreads();
            }        
    }
    flow_share[tid] = temp[tid + n * out];
    __syncthreads();

    

    int j;
//计算不同流有多少个数据包
    if(tid > 0)
    {
        if(flow_share[tid] != flow_share[tid - 1])
        {
            j = flow_share[tid];
            flag_share[j] = tid;

        }
    }
    __syncthreads();
   
   j = flow_share[tid];
   feature_share[tid] = feature_share[tid] - flag_share[j];
    __syncthreads();

    a[tid] = hash_share[tid];
    b[tid] = feature_share[tid];

}

int main(int argc, char** argv)
{
    int n = 1024;
    int array[n];
    int back[n];

    for(int i = 0 ; i < n; i++)
    {
        array[i] = rand()%64;
        back[i] = array[i];
        printf("%d ",array[i]);
    }
    printf("\n\n");

    size_t nbytes = n * sizeof(int);
    int* Garray = NULL;
    hipMalloc((void**)&Garray,nbytes);
  
    int* Back = NULL;
    hipMalloc((void**)&Back,nbytes);
    
    hipMemcpy(Garray, array, nbytes, hipMemcpyHostToDevice);

    // 执行内核函数
    warpfun<<<1, n>>>(Garray,Back, n);
    hipDeviceSynchronize(); // 确保内核执行完成
    
    hipMemcpy(array, Garray, nbytes, hipMemcpyDeviceToHost);
    hipMemcpy(back, Back, nbytes, hipMemcpyDeviceToHost);
    
    
    for(int i = 0 ; i < n; i++)
    {
        printf("%d ",array[i]);
    }
    printf("\n\n");
 
    for(int i = 0 ; i < n; i++)
    {
        //if((back[i]-back[i+1]) > 0)
            printf("%d ",back[i]);
    }    
    printf("\n\n");
    for(int i = 0 ; i < n-1; i++)
    {
        if((back[i]-back[i+1]) > 0)
            printf("%d ",back[i]);
    }
    printf("\n\n");

    hipFree(Garray);
    hipFree(Back);

    return EXIT_SUCCESS;
}
